#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void hello_kernel(char *odata, int num) {
    char hello_str[480] = "#######################################\n  _    _                  _            \n | |  | |                | |           \n | |__| | ___ _ __   __ _| | ___  ___  \n |  __  |/ _ \\ '_ \\ / _` | |/ _ \\/ _ \\ \n | |  | |  __/ | | | (_| | |  __/  __/ \n |_|  |_|\\___|_| |_|\\__, |_|\\___|\\___| \n                     __/ |             \n                    |___/              \n+++++++++++++++++++++++++++++++++++++++\n:::::::::::::::::::::::::::::::::::::::\n";
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num)
        odata[idx] = hello_str[idx];
}

int main(void) {
    char *h_data, *d_data;
    const int strlen = 480;
    size_t strsize = strlen * sizeof(char);
    h_data = (char *) malloc(strsize);
    memset(h_data, 0, strlen);
    hipMalloc((void **) &d_data, strsize);
    hipMemcpy(d_data, h_data, strsize, hipMemcpyHostToDevice);
    int blocksize = 8;
    int nblock = strlen/blocksize + (strlen % blocksize == 0 ? 0 : 1);

    hello_kernel<<<nblock,blocksize>>>(d_data, strlen);

    hipMemcpy(h_data, d_data, sizeof(char)*strlen, hipMemcpyDeviceToHost);
    printf("%s\n", h_data);

    free(h_data);
    hipFree(d_data);
}